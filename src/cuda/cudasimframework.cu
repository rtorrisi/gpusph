#include "hip/hip_runtime.h"
/*  Copyright 2014 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef _CUDASIMFRAMEWORK_H
#define _CUDASIMFRAMEWORK_H

#include "simframework.h"

#include "predcorr_alloc_policy.h"

#include "simflags.h"
#include "textures.cuh"

#include "sph_core.cu"
#include "phys_core.cu"
#include "buildneibs.cu"
#include "geom_core.cu"
#include "boundary_conditions.cu"
#include "euler.cu"
#include "forces.cu"
#include "visc.cu"
#include "post_process.cu"
#include "option_range.h"

using namespace std;

using namespace std;

// This class holds the implementation and interface of CUDASimFramework,
// the CUDA simulation framework for GPUPSH. (In fact, the only simulation
// framework in GPUSPH, presently).

// The CUDASimFramework is a template class depending on KernelType, ViscSpec,
// BoundaryType, Periodicity and simulation flags, in order to allow concrete
// instantiation of only the needed specializations of the appropriate engines.

// To allow the user to specify any (or none, or all) of the template parameters,
// in any order, we separate the implementation from the user-visible class that
// instantiates it. This is not strictly necessary, but it makes the code more
// readable by separating the optional, named template arguments management
// from the implementation of the actual framework.


/* CUDABoundaryConditionsSelector */

// We begin with an auxiliary functor to select a BoundaryConditionsEngine,
// which currently simply selects the CUDABoundaryConditionsEngine in the case of
// SA_BOUNDARY BoundaryType, and returns NULL otherwise.

// General case
template<KernelType kerneltype, typename ViscSpec,
	BoundaryType boundarytype, flag_t simflags>
struct CUDABoundaryConditionsSelector
{
	typedef CUDABoundaryConditionsEngine<kerneltype, ViscSpec, boundarytype, simflags> BCEtype;
	static BCEtype* select()
	{ return NULL; } // default, no BCE
};

// SA_BOUNDARY specialization
template<KernelType kerneltype, typename ViscSpec, flag_t simflags>
struct CUDABoundaryConditionsSelector<kerneltype, ViscSpec, SA_BOUNDARY, simflags>
{
	typedef CUDABoundaryConditionsEngine<kerneltype, ViscSpec, SA_BOUNDARY, simflags> BCEtype;
	static BCEtype* select()
	{ return new BCEtype(); } // TODO FIXME when we have proper BCEs
};

/// Some combinations of frameworks for kernels are invalid/
/// unsupported/untested and we want to prevent the user from
/// using them, by (1) catching the error as soon as possible
/// during compilation and (2) give an error message that is
/// as descriptive as possible (non-trivial with C++).
/// Point (2) is particularly hard to realize with nvcc because
/// it doesn't print out the actual line with the error, so we
/// need some indirection; we achieve this by making the
/// CUDASimFramework subclass a template class InvalidOptionCombination
/// whose instantiation will fail in case of invalid option combinations;
/// this failure is due to trying to subclass an IncompleteType class
/// which is not defined except in the not invalid case.
/// nvcc will then show the error for InvalidOptionCombination, which
/// is hopefully descriptive enough for users.
template<bool invalid>
class IncompleteType;

template<>
class IncompleteType<false>
{};

template<bool invalid>
class InvalidOptionCombination : IncompleteType<invalid>
{};

/* CUDASimFrameworkImpl */

// Here we define the implementation for the CUDASimFramework. The use of *Impl is
// to allow the user-facing CUDASimFramework to be the one that allows named and optional
// template parameters

template<
	KernelType _kerneltype,
	SPHFormulation _sph_formulation,
	DensityDiffusionType _densitydiffusiontype,
	RheologyType _rheologytype,
	TurbulenceModel _turbmodel,
	ComputationalViscosityType _compvisc,
	ViscousModel _viscmodel,
	AverageOperator _viscavgop,
	LegacyViscosityType _legacyvisctype,
	BoundaryType _boundarytype,
	Periodicity _periodicbound,
	flag_t _simflags,
	bool _is_const_visc = (_legacyvisctype == KINEMATICVISC) || (
		IS_SINGLEFLUID(_simflags) &&
		(_rheologytype == NEWTONIAN) &&
		(_turbmodel != KEPSILON)
	),
	bool invalid_combination =
		// Currently, we consider invalid only the case
		// of SA_BOUNDARY

		// TODO extend to include all unsupported/untested combinations for other boundary conditions

		(_legacyvisctype == KINEMATICVISC && IS_MULTIFLUID(_simflags)) || // kinematicvisc model only made sense for single-fluid
		(_turbmodel == KEPSILON && _boundarytype != SA_BOUNDARY) || // k-epsilon only supported in SA currently
		(_boundarytype == SA_BOUNDARY && (
			// viscosity
			_viscmodel != MORRIS			||	// untested
			_viscavgop != ARITHMETIC		||	// untested
			_turbmodel == SPS			||	// untested
			_turbmodel == ARTIFICIAL		||	// untested (use is discouraged, use density diffusion instead)
			// kernel
			! (_kerneltype == WENDLAND)		||	// only the Wendland kernel is allowed in SA_BOUNDARY
												// all other kernels would require their respective
												// gamma and grad gamma formulation
			// formulation
			_sph_formulation == SPH_GRENIER	||	// multi-fluid is currently not implemented
			// flags
			_simflags & ENABLE_XSPH			||	// untested
			_simflags & ENABLE_DEM			||	// not implemented (flat wall formulation is in an old branch)
			(_simflags & ENABLE_INLET_OUTLET && !(_simflags & ENABLE_DENSITY_SUM)) ||
												// inlet outlet works only with the summation density
			(_simflags & ENABLE_DENSITY_SUM && _simflags & ENABLE_GAMMA_QUADRATURE)
												// enable density sum only works with the dynamic equation for gamma,
												// so gamma quadrature must be disabled
		)
	) || (
	!(_boundarytype == SA_BOUNDARY) && _simflags & ENABLE_DENSITY_SUM
												// density sum is untested with boundary conditions other than SA
	) || (
	// For Español & Revenga, currently only support Newtonian fluids with
	// arithmetic averaging and dynamic computational viscosity; to support
	// other averaging operators or kinematic viscosities, we should first define
	// how averaging is achieved (i.e. on the coefficients as a whole or for the
	// individual viscosities); for non-Newtonian fluids we need a way to compute
	// the bulk viscosities
	_viscmodel == ESPANOL_REVENGA && _rheologytype != NEWTONIAN
	)
>
class CUDASimFrameworkImpl : public SimFramework,
	private InvalidOptionCombination<invalid_combination>
{
public:
	static const KernelType kerneltype = _kerneltype;
	static const SPHFormulation sph_formulation = _sph_formulation;
	static const DensityDiffusionType densitydiffusiontype = _densitydiffusiontype;

	static const RheologyType rheologytype = _rheologytype;
	static const TurbulenceModel turbmodel = _turbmodel;
	static const ComputationalViscosityType compvisc = _compvisc;
	static const ViscousModel viscmodel = _viscmodel;
	// Grenier used to assume harmonic averaging regardless of the specification. Today we support
	// overriding the choice, but for backwards compatibility we should still assume harmonic averaging
	// when legacy viscous specifications have been used
	static const AverageOperator viscavgop = (
		((_sph_formulation == SPH_GRENIER) && // when using Grenier's formulation
		(_legacyvisctype != INVALID_VISCOSITY)) ? // was there a legacy specification?
		AverageOperator::HARMONIC : // yes, assume harmonic averaging 
		_viscavgop // no, take what the user gave us
	);
	static const bool is_const_visc = _is_const_visc;

	using ViscSpec = FullViscSpec<_rheologytype, _turbmodel, _compvisc,
	      _viscmodel, viscavgop, _simflags, _is_const_visc>;

	static const BoundaryType boundarytype = _boundarytype;
	static const Periodicity periodicbound = _periodicbound;
	static const flag_t simflags = _simflags;

public:
	CUDASimFrameworkImpl() : SimFramework()
	{
		m_neibsEngine = new CUDANeibsEngine<sph_formulation, boundarytype, periodicbound, true>();
		m_integrationEngine = new CUDAPredCorrEngine<sph_formulation, boundarytype, kerneltype, ViscSpec, simflags>();
		m_viscEngine = new CUDAViscEngine<ViscSpec, kerneltype, boundarytype, simflags>();
		m_forcesEngine = new CUDAForcesEngine<kerneltype, sph_formulation, densitydiffusiontype, ViscSpec, boundarytype, simflags>();
		m_bcEngine = CUDABoundaryConditionsSelector<kerneltype, ViscSpec, boundarytype, simflags>::select();

		// TODO should be allocated by the integration scheme
		m_allocPolicy = make_shared<PredCorrAllocPolicy>();

		m_simparams = new SimParams(this);
	}

protected:
	AbstractFilterEngine* newFilterEngine(FilterType filtertype, int frequency)
	{
		switch (filtertype) {
		case SHEPARD_FILTER:
			return new CUDAFilterEngine<SHEPARD_FILTER, kerneltype, boundarytype>(frequency);
		case MLS_FILTER:
			return new CUDAFilterEngine<MLS_FILTER, kerneltype, boundarytype>(frequency);
		case INVALID_FILTER:
			throw runtime_error("Invalid filter type");
		}
		throw runtime_error("Unknown filter type");
	}

	AbstractPostProcessEngine* newPostProcessEngine(PostProcessType pptype, flag_t options=NO_FLAGS)
	{
		switch (pptype) {
		case VORTICITY:
			return new CUDAPostProcessEngine<VORTICITY, kerneltype, boundarytype, simflags>(options);
		case TESTPOINTS:
			return new CUDAPostProcessEngine<TESTPOINTS, kerneltype, boundarytype, simflags>(options);
		case SURFACE_DETECTION:
			return new CUDAPostProcessEngine<SURFACE_DETECTION, kerneltype, boundarytype, simflags>(options);
		case FLUX_COMPUTATION:
			return new CUDAPostProcessEngine<FLUX_COMPUTATION, kerneltype, boundarytype, simflags>(options);
		case CALC_PRIVATE:
			return new CUDAPostProcessEngine<CALC_PRIVATE, kerneltype, boundarytype, simflags>(options);
		case INVALID_POSTPROC:
			throw runtime_error("Invalid filter type");
		}
		throw runtime_error("Unknown filter type");
	}

};

/* CUDASimFramework user-facing interface */

// We want to allow the user to create a CUDASimFramework by omitting any of the template
// parameters, and to override them in any order. For example, if the user wants to
// override only the kernel and the periodicity, and to enable XSPH, they should be able to
// write something like:
//
//	m_simframework = new CUDASimFramework<
//		withKernel<WENDLAND>,
//		withFlags<ENABLE_XSPH | ENABLE_DEM>,
//		withPeriodicity<PERIODIC_X>
//	>();
//
// NOTE: the withFlags<> will override the default flags, not add to them,
// so in case of flag override, the default ones should be included manually.
// As an alternative, a class that adds to the defaults is provided too.

// TODO we may want to put the implementation of the named template options into its own
// header file.

// To get to the named, optional parameter template API we will need a couple of auxiliary
// classes. The main mechanism is essentially inspired by the named template arguments
// mechanism shown in http://www.informit.com/articles/article.aspx?p=31473 with some
// additions to take into account that our template arguments are not typenames, but
// values of different types, and to allow inheritance from previous arguments selectors.

// The first auxiliary class is TypeValue: a class template to carry a value and its type:
// this will be used to specify the default values for the parameters, as well
// as to allow their overriding by the user. It is needed because we want to
// allow parameters to be specified in any order, and this means that we need a
// common 'carrier' for our specific types.

template<typename T, T _val>
struct TypeValue
{
	typedef T type;
	static const T value = _val;
	constexpr operator T() const { return _val; }; // allow automatic conversion to the type
};

// We will rely on multiple inheritance to group the arguments, and we need to be
// able to specify the same class multiple times (which is forbidden by the standard),
// so we will wrap the type in a "multiplexer":

template<typename T, int idx>
struct MultiplexSubclass : virtual public T
{};

// Template arguments are collected into this class: it will subclass
// all of the template arguments, that must therefore have a common base class
// (see below), and uses the multiplexer class above in case two ore more arguments
// are actually the same class. The number of supported template arguments
// should match that of the CUDASimFramework

template<typename Arg1, typename Arg2, typename Arg3,
	typename Arg4, typename Arg5, typename Arg6,
	typename Arg7, typename Arg8, typename Arg9,
	typename Arg10, typename Arg11, typename Arg12>
struct ArgSelector :
	virtual public MultiplexSubclass<Arg1,1>,
	virtual public MultiplexSubclass<Arg2,2>,
	virtual public MultiplexSubclass<Arg3,3>,
	virtual public MultiplexSubclass<Arg4,4>,
	virtual public MultiplexSubclass<Arg5,5>,
	virtual public MultiplexSubclass<Arg6,6>,
	virtual public MultiplexSubclass<Arg7,7>,
	virtual public MultiplexSubclass<Arg8,8>,
	virtual public MultiplexSubclass<Arg9,9>,
	virtual public MultiplexSubclass<Arg10,10>,
	virtual public MultiplexSubclass<Arg11,11>,
	virtual public MultiplexSubclass<Arg12,12>
{};

// Now we set the defaults for each argument
struct TypeDefaults
{
	typedef TypeValue<KernelType, WENDLAND> Kernel;
	typedef TypeValue<SPHFormulation, SPH_F1> Formulation;
	typedef TypeValue<DensityDiffusionType, DENSITY_DIFFUSION_NONE> DensityDiffusion;
	typedef TypeValue<RheologyType, INVISCID> Rheology;
	typedef TypeValue<TurbulenceModel, ARTIFICIAL> Turbulence;
	typedef TypeValue<ComputationalViscosityType, KINEMATIC> ComputationalViscosity;
	typedef TypeValue<ViscousModel, MORRIS> ViscModel;
	typedef TypeValue<AverageOperator, ARITHMETIC> ViscAveraging;
	typedef TypeValue<LegacyViscosityType, INVALID_VISCOSITY> LegacyViscType;
	typedef TypeValue<BoundaryType, LJ_BOUNDARY> Boundary;
	typedef TypeValue<Periodicity, PERIODIC_NONE> Periodic;
	typedef TypeValue<flag_t, DEFAULT_FLAGS> Flags;
};

// The user-visible name template parameters will all subclass TypeDefaults,
// and override specific typedefs
// NOTE: inheritance must be virtual so that there will be no resolution
// ambiguity.
// NOTE: in order to allow the combination of a named parameter struct with
// an existing (specific) ArgSelector, we allow them to be assigned a different
// parent, in order to avoid resolution ambiguity in constructs such as:
// ArgSelector<OldArgSelector, formulation<OTHER_FORMULATION> >

// No override: these are the default themselves
struct DefaultArg : virtual public TypeDefaults
{};

//! A structure that maps to the selector for the specific type
template<typename Option, Option value>
struct selector_for;

#define DEFINE_ARGSELECTOR(selector, SelectorType, ArgName) \
template<SelectorType value__, typename ParentArgs=TypeDefaults> \
struct selector : virtual public ParentArgs \
{ \
	typedef TypeValue<SelectorType, value__> ArgName; \
	template<typename NewParent> struct reparent : \
		virtual public selector<value__, NewParent> {}; \
}; \
template<SelectorType value> \
struct selector_for<SelectorType, value> : virtual public selector<value> \
{}


// Kernel override
DEFINE_ARGSELECTOR(kernel, KernelType, Kernel);

// Formulation override
DEFINE_ARGSELECTOR(formulation, SPHFormulation, Formulation);

// Density diffusion override
DEFINE_ARGSELECTOR(densitydiffusion, DensityDiffusionType, DensityDiffusion);

// Rheology override
DEFINE_ARGSELECTOR(rheology, RheologyType, Rheology);

// Turbulence model override
DEFINE_ARGSELECTOR(turbulence_model, TurbulenceModel, Turbulence);

// ComputationalViscosity override
DEFINE_ARGSELECTOR(computational_visc, ComputationalViscosityType, ComputationalViscosity);

// ViscousModel override
DEFINE_ARGSELECTOR(visc_model, ViscousModel, ViscModel);

// AverageOperator override
DEFINE_ARGSELECTOR(visc_average, AverageOperator, ViscAveraging);

template<LegacyViscosityType visctype, typename ParentArgs=TypeDefaults>
struct viscosity : virtual public ParentArgs
{
	// propagate the information about the fact that the user
	// specified the given legacy type
	typedef TypeValue<LegacyViscosityType, visctype> LegacyViscType;

	// set the corresponding viscous model parameters
	using Spec = typename ConvertLegacyVisc<visctype>::type;
	typedef TypeValue<RheologyType, Spec::rheologytype> Rheology;
	typedef TypeValue<TurbulenceModel, Spec::turbmodel> Turbulence;
	typedef TypeValue<ComputationalViscosityType, Spec::compvisc> ComputationalViscosity;
	typedef TypeValue<ViscousModel, Spec::viscmodel> ViscModel;
	typedef TypeValue<AverageOperator, Spec::avgop> ViscAveraging;

	template<typename NewParent> struct reparent :
		virtual public viscosity<visctype, NewParent> {};
};

// Boundary override
DEFINE_ARGSELECTOR(boundary, BoundaryType, Boundary);

// Periodic override
DEFINE_ARGSELECTOR(periodicity, Periodicity, Periodic);

#if 0
// Flags override
// These are disabled because problems should only use
// add_flags<> and disable_flags<>, in order to avoid issues
// when new default flags get introduced for backwards compatibility
DEFINE_ARGSELECTOR(flags, flag_t, Flags);
#endif

// Add flags: this is an override that adds the new simflags
// to the ones of the parent.
template<flag_t simflags, typename ParentArgs=TypeDefaults>
struct add_flags : virtual public ParentArgs
{
	typedef TypeValue<flag_t, ParentArgs::Flags::value | simflags> Flags;

	template<typename NewParent> struct reparent :
		virtual public add_flags<simflags, NewParent> {};
};

// Disable flags: this is an override that removes the given simflags
// from the ones of the parent
template<flag_t simflags, typename ParentArgs=TypeDefaults>
struct disable_flags : virtual public ParentArgs
{
	typedef TypeValue<flag_t, DISABLE_FLAGS(ParentArgs::Flags::value, simflags)> Flags;

	template<typename NewParent> struct reparent :
		virtual public add_flags<simflags, NewParent> {};
};

/// Our CUDASimFramework is actualy a factory for CUDASimFrameworkImpl*,
/// generating one when assigned to a SimFramework*. This is to allow us
/// to change the set of options at runtime without setting up/tearing down
/// the whole simframework every time an option is changed (setting up/tearing
/// down the factory itself is much cheaper as there is no associated storage, so
/// it's mostly just compile-time juggling).
template<
	typename Arg1 = DefaultArg,
	typename Arg2 = DefaultArg,
	typename Arg3 = DefaultArg,
	typename Arg4 = DefaultArg,
	typename Arg5 = DefaultArg,
	typename Arg6 = DefaultArg,
	typename Arg7 = DefaultArg,
	typename Arg8 = DefaultArg,
	typename Arg9 = DefaultArg,
	typename Arg10 = DefaultArg,
	typename Arg11 = DefaultArg,
	typename Arg12 = DefaultArg>
class CUDASimFramework {
	/// The collection of arguments for our current setup
	typedef ArgSelector<Arg1, Arg2, Arg3, Arg4, Arg5, Arg6,
		Arg7, Arg8, Arg9, Arg10, Arg11, Arg12> Args;

	/// Comfort static defines
	static const KernelType kerneltype = Args::Kernel::value;
	static const SPHFormulation sph_formulation = Args::Formulation::value;
	static const DensityDiffusionType densitydiffusiontype = Args::DensityDiffusion::value;

	static const RheologyType rheologytype = Args::Rheology::value;
	static const TurbulenceModel turbmodel = Args::Turbulence::value;
	static const ComputationalViscosityType compvisc = Args::ComputationalViscosity::value;
	static const ViscousModel viscmodel = Args::ViscModel::value;
	static const AverageOperator viscavgop = Args::ViscAveraging::value;

	static const BoundaryType boundarytype = Args::Boundary::value;
	static const Periodicity periodicbound = Args::Periodic::value;
	static const flag_t simflags = Args::Flags::value;

	/// The CUDASimFramework implementation of the current setup
	typedef CUDASimFrameworkImpl<
			kerneltype,
			sph_formulation,
			densitydiffusiontype,
			rheologytype,
			turbmodel,
			compvisc,
			viscmodel,
			viscavgop,
			Args::LegacyViscType::value,
			boundarytype,
			periodicbound,
			simflags> CUDASimFrameworkType;

	/// A comfort auxiliary class that overrides Args (the current setup)
	/// with the Extra named option
	template<typename Extra> struct Override :
		virtual public Args,
		virtual public Extra::template reparent<Args>
	{};

	/// A method to produce a new factory with an overridden parameter
	template<typename Extra>
	CUDASimFramework< Override<Extra> > extend() {
		return CUDASimFramework< Override<Extra> >();
	}

public:
	/// Conversion operator: this produces the actual implementation of the
	/// simframework
	operator SimFramework *()
	{
		// return the intended framework
		return new CUDASimFrameworkType();
	}

	/// Runtime selectors.

	/// Note that they must return a SimFramework* because otherwise the type
	/// returned would depend on the runtime selection, which is not possible.
	/// As a result we cannot chain runtime selectors, and must instead provide
	/// further runtime selectors with multiple (pairs of) overrides

	/// Select an override only if a boolean option is ture
	template<typename Extra>
	SimFramework * select_options(bool selector, Extra)
	{
		if (selector)
			return extend<Extra>();
		return *this;
	}

	/// Select a run-time override based on an option value
	template<typename Option, Option check = option_range<Option>::min>
	enable_if_t<option_range<Option>::defined && is_in_range_t<Option, check>::value, SimFramework *>
	select_options(Option selector)
	{
		if (selector == check)
			return extend< selector_for<Option, check> >();
		return select_options<Option, Option(check+1)>(selector);
	}

	template<typename Option, Option check>
	enable_if_t<not is_in_range_t<Option, check>::value, SimFramework *>
	select_options(Option selector)
	{
		throw runtime_error("invalid selector value");
	}

	/// Chained selectors (for multiple overrides)
	template<typename Extra, typename ...Rest>
	SimFramework * select_options(bool selector, Extra, Rest...rest)
	{
		if (selector)
			return extend<Extra>().select_options(rest...);
		return this->select_options(rest...);
	}

	/// Chained selectors (for multiple overrides)
	template<typename Option, Option check = option_range<Option>::min, typename ...Rest>
	enable_if_t<option_range<Option>::defined && is_in_range_t<Option, check>::value, SimFramework *>
	select_options(Option selector, Rest...rest)
	{
		if (selector == check)
			return extend< selector_for<Option, check> >().select_options(rest...);
		return select_options<Option, Option(check+1), Rest...>(selector, rest...);
	}

	template<typename Option, Option check, typename ...Rest>
	enable_if_t<not is_in_range_t<Option, check>::value, SimFramework *>
	select_options(Option selector, Rest...rest)
	{
		throw runtime_error("invalid selector value");
	}

};

#endif

/* vim: set ft=cuda sw=4 ts=4 : */
